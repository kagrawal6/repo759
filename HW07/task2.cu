#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

int main(int argc, char *argv[]) {
   
    
    unsigned int N = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);
    
    // Create and fill an array of length N with random numbers in the range [-1,1] on the host
    float *h_input = (float*)malloc(N * sizeof(float));
    
    //random
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
    
    for (unsigned int i = 0; i < N; i++) {
        h_input[i] = dis(gen);
    }
    
    // device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    
    // Calculate the number of blocks 
    unsigned int num_blocks = (N + 2 * threads_per_block - 1) / (2 * threads_per_block);
    hipMalloc(&d_output, num_blocks * sizeof(float));
    
    
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start 
    hipEventRecord(start);
    
    // Call
    reduce(&d_input, &d_output, N, threads_per_block);
    
    // Stop 
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate  time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result
    float result;
    hipMemcpy(&result, d_input, sizeof(float), hipMemcpyDeviceToHost);
    
    // Print 
    printf("%f\n", result);
    printf("%f\n", milliseconds);
    
    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_input);
    
    return 0;
}