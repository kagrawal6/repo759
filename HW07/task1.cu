#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <cstdio>      
#include <cstdlib>     
#include <iostream>    
#include <string>     

int main(int argc, char *argv[]) {
   

    
    unsigned int n = std::stoi(argv[1]);
    unsigned int block_dim = (argc > 2) ? std::stoi(argv[2]) : 16;

    // Limit matrix size to 2^14
    if (n > (1u << 14)) {
        std::cerr << "matrix size must be <= 2^14\n";
        return 1;
    }

   
    hipEvent_t start, stop;
    float milliseconds = 0.0f;

    //  Integer 
    {
        //  int matrices
        int *A, *B, *C;
        hipMallocManaged(&A, n * n * sizeof(int));
        hipMallocManaged(&B, n * n * sizeof(int));
        hipMallocManaged(&C, n * n * sizeof(int));

        
        for (unsigned int i = 0; i < n; ++i) {
            for (unsigned int j = 0; j < n; ++j) {
                A[i * n + j] = (i + j) % 10;
                B[i * n + j] = (i * j) % 10;
            }
        }

        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Execute integer 
        matmul_1(A, B, C, n, block_dim);

        
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Measure elapsed time
        hipEventElapsedTime(&milliseconds, start, stop);

        // Output first element, last element, and runtime
        std::cout << C[0] << "\n"
                  << C[n * n - 1] << "\n"
                  << milliseconds << "\n";

        // Clean up events and memory
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

    // Float
    {
        // Allocate unified memory for float matrices
        float *A, *B, *C;
        hipMallocManaged(&A, n * n * sizeof(float));
        hipMallocManaged(&B, n * n * sizeof(float));
        hipMallocManaged(&C, n * n * sizeof(float));

        // Fill A and B with fractional patterns
        for (unsigned int i = 0; i < n; ++i) {
            for (unsigned int j = 0; j < n; ++j) {
                A[i * n + j] = ((i + j) % 10) / 10.0f;
                B[i * n + j] = ((i * j) % 10) / 10.0f;
            }
        }

        // Create and record start event
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Execute float tiled matmul
        matmul_2(A, B, C, n, block_dim);

        // Record stop event and wait for kernel finish
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Measure elapsed time
        hipEventElapsedTime(&milliseconds, start, stop);

        // Output first element, last element, and runtime
        std::cout << C[0] << "\n"
                  << C[n * n - 1] << "\n"
                  << milliseconds << "\n";

        // Clean up events and memory
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

    // Double
    {
        // Allocate unified memory for double matrices
        double *A, *B, *C;
        hipMallocManaged(&A, n * n * sizeof(double));
        hipMallocManaged(&B, n * n * sizeof(double));
        hipMallocManaged(&C, n * n * sizeof(double));

        // Fill A and B with fractional patterns
        for (unsigned int i = 0; i < n; ++i) {
            for (unsigned int j = 0; j < n; ++j) {
                A[i * n + j] = ((i + j) % 10) / 10.0;
                B[i * n + j] = ((i * j) % 10) / 10.0;
            }
        }

        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        
        matmul_3(A, B, C, n, block_dim);

       
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Measure elapsed time
        hipEventElapsedTime(&milliseconds, start, stop);

        // Output first element, last element, and runtime
        std::cout << C[0] << "\n"
                  << C[n * n - 1] << "\n"
                  << milliseconds << "\n";

        
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

    return 0;
}
